
#include <hip/hip_runtime.h>
#include<chrono>
#include<iostream>
#include<math.h>

// Define function here /
#define f(x) 4*(1/(1+pow(x,2)))

using namespace std;
int main()
{
	float lower, upper, integration = 0.0, stepSize, k;
	int i, subInterval;

	// Input /
	lower = 0;
	upper = 1;
	cout << "Enter number of sub intervals: ";
	cin >> subInterval;

	// Calculation /

	// Finding step size /
	stepSize = (upper - lower) / subInterval;

	// Finding Integration Value /
	integration = f(lower) + f(upper);
	auto start = std::chrono::high_resolution_clock::now();
	for (i = 1; i <= subInterval - 1; i++)
	{
		k = lower + i * stepSize;
		integration = integration + 2 * (f(k));
	}

	integration = integration * stepSize / 2;
	auto finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = finish - start;

	cout << endl << "Required value of integration is: " << integration;
	cout << endl << "Duration of calculations is: " << elapsed.count();

	return 0;
}