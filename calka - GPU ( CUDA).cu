
#include <hip/hip_runtime.h>
#include<iostream>
#include<chrono>

 __global__ void integratorKernel(float *a, int N)
{int idx = blockIdx.x * blockDim.x + threadIdx.x;
float x = (float)idx / (float)N;

if(idx < N) 
a[idx] = 1/(1+x*x);
}

float cudaIntegrate(int N)
{
// size of the arrays in bytessize_t 
float size = N * sizeof(float);

// allocate array on host and device
float* a_h = (float *)malloc(size);
float* a_d; hipMalloc((void **) &a_d, size);

// do calculation on device
int block_size = 256;
int n_blocks = N/block_size + (N % block_size == 0 ? 0:1);
integratorKernel <<< n_blocks, block_size >>> (a_d, N);

// copy results from device to host
hipMemcpy(a_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);
// add up resultsfloat 
float sum = 0;
for (int i=0; i < N; i++) 
sum += a_h[i];
sum *= 1.0 / (float)N;;

// clean up
free(a_h); 
hipFree(a_d);
return sum;
}

int main(int argc, char** argv){
    auto start = std::chrono::high_resolution_clock::now();

float x = cudaIntegrate(10000000000);
x=x*4;
auto finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = finish - start;

printf("Przyblizona wartosc calki dla GPU wynosi %f \n",x );
printf("Przyblizona czas calki wynosi %f",elapsed.count());

}



