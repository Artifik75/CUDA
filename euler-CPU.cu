
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include<chrono>
 

int main(void) {

    double gammaEulera = 0.; 
    double N = 1000000;;  

    auto start = std::chrono::high_resolution_clock::now();

        for (int i = 1; i < N; i++)
        gammaEulera = gammaEulera + (1. / (double)i); 

    gammaEulera = gammaEulera - log(N);

  auto finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = finish - start;

    

    printf("Przyblizona wartosc eulera CPU wynosi %f \n",gammaEulera );
    printf("Czas wynosi %f",elapsed.count());
    return 0;
}

 